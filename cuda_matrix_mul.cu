#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <conio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void mul_matrix_on_gpu( float* a, float *b, float *c, int N )
{
	float sum=0;
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockDim.y*blockIdx.y+threadIdx.y;
	for(int i=0;i<N;i++){
	if(x<N && y<N){
	int indexa = x*N+i;
	int indexb = i*N+y;
	sum+=a[indexa]*b[indexb];}
	}
	c[x*N+y]=sum;
}

int main() 
{
	int N;
	printf("\nEnter the no: of rows/columns in the matrix : ");
scanf("%d",&N);
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	printf("\nEnter the 1st matrix\n");
	for ( int i = 0; i < N*N; ++i ) {
		scanf("%f",&a[i]); }

	printf("\nEnter the 2nd matrix\n");
	for ( int i = 0; i < N*N; ++i ) {
		scanf("%f",&b[i]); }

	float *ad, *bd, *cd;
	const int size = N*N*sizeof(float);

	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );
	hipMalloc( (void**)&cd, size );


	hipMemcpy( ad, a, size, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice );

	int blocksize;
	if(N>2)
	for(blocksize=1;N%blocksize!=0;++blocksize);
	else blocksize=1;
	printf("\nBlock Size = %d\n",blocksize);

	dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( N/dimBlock.x, N/dimBlock.y );

	mul_matrix_on_gpu<<<dimGrid, dimBlock>>>( ad, bd, cd, N );
        
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );
	
for(int i=0;i<N;i++){
for(int j=0;j<N;j++)
printf(" %f ",a[i*N + j]);
printf("\n");}

printf("\n + \n");

for(int i=0;i<N;i++){
for(int j=0;j<N;j++)
printf(" %f ",b[i*N + j]);
printf("\n");}
	
printf("\nPRODUCT \n");

for(int i=0;i<N;i++){
for(int j=0;j<N;j++)
printf(" %f ",c[i*N + j]);
printf("\n");}

        hipFree( ad ); hipFree( bd ); hipFree( cd );
	delete[] a; delete[] b; delete[] c;
	getch();
	return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
// Copyright (c) 2014 Godly T.Alias
//
// This is a free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation,
// either version 3 of the License, or (at your option) any later version.

#include<iostream>
#include<conio.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<direct.h>
using namespace std;


int n1,n2,perm;
struct mapping
{
int map_ver;
float state;
int classid; };


float *g1,*g2;
int node,w_node;
int tmp_count;
mapping *map_g;

__device__ void max_heapify(float *a,mapping *pos, int i, int n)
{
    int j, temps;
    float temp;
    temps = pos[i].map_ver;
    temp = a[temps];
    j = 2*i;
    while (j <= n)
    {
        if (j < n && a[pos[j+1].map_ver] > a[pos[j].map_ver])
            j = j+1;
        if (temp > a[pos[j].map_ver])
            break;
        else if (temp <= a[pos[j].map_ver])
        {
            pos[j/2].map_ver = pos[j].map_ver;
            j = 2*j;
        }
    }
    pos[j/2].map_ver = temps;
    return;
}

__device__ void heapsort(float *a,mapping *pos, int end)
{
    int i, temps;
    for (i = end; i >= 2; i--)
    {
        temps = pos[i].map_ver;
        pos[i].map_ver = pos[1].map_ver;
        pos[1].map_ver = temps;
        max_heapify(a,pos, 1, i - 1);
    }
}

__device__ void build_maxheap(float *a,mapping *pos, int end)
{
    int i;
    for(i = end/2; i >= 1; i--)
    {
        max_heapify(a,pos, i, end);
    }
}

bool adj_mat_map(float *a1, float *a2)
{
int i,j;
for(i=0;i<node;i++)
 for(j=0;j<node;j++)
  if(a1[map_g[i].map_ver*node+map_g[j].map_ver]!=a2[map_g[1*node+i].map_ver*node+map_g[1*node+j].map_ver])
   return false;
return true;
}


int isotest(int p1_init_node,int p2_init_node,float *a1,float *a2)
{
    char filename[40];
    sprintf(filename,"../graphiso/map_0_%d",p1_init_node);
    FILE *read1 = fopen(filename,"r");
    sprintf(filename,"../graphiso/map_1_%d",p2_init_node);
    FILE *read2 = fopen(filename,"r");
while(!feof(read1)){
for(int i=0;i<node;i++)
 fscanf(read1,"%d",&map_g[i].map_ver);

while(!feof(read2)){
for(int j=0;j<node;j++)
 fscanf(read2,"%d",&map_g[1*node+j].map_ver);

  if(adj_mat_map(a1,a2))
 { fclose(read1);
   fclose(read2);
   return 2; } }
      
 fseek(read2,0,SEEK_SET);
    }
fclose(read1);
fclose(read2);
 return 0;
}


//returns the initial state distribution vector
__device__ void istate_dibn_vec(float* init_state, int i,int n)
{
for(int j=0;j<n;j++)
 if(j==i)
  init_state[j]=1.0;
 else
  init_state[j]=0.0;
}

//computes the product of matrices m1 & m2 and write the result in res matrix
__device__ void matrix_prod(float *res,float *m1,int c1,float *m2,int r2,int c2)
{
float y,t,c;
if(c1==r2){
 for(int j=0;j<c2;j++){
 res[j]=0;
 c=0.0;
  for(int k=0;k<c1;k++){
//kahan summation to avoid precision lose
  y=(m1[k]*m2[k*c2+j])-c;
  t=res[j]+y;
  c = (t-res[j])-y;
  res[j]=t;}
 }
}}

//calculates the probability propogation matrix for the initial state initstate
__global__ void prob_prop_matrix(int graph_id, float *g, int n, mapping *map_g,float *row_mat,float *row_mat_copy)
{
	int initstate = blockIdx.x*blockDim.x+threadIdx.x;
	int ptr = (graph_id+initstate)*n;
	 for(int i=0;i<n;i++)
        {
        map_g[ptr+i].map_ver=i;
        map_g[ptr+i].state=-1.0;
        map_g[ptr+i].classid=0;
        }

bool flag=true;
int start,end,j,temp,classptr;
float temps;

//writes the initial state vector to the row_mat
istate_dibn_vec(row_mat,initstate,n);
classptr=1;
for(int i=0;flag && i<((2*n)-1);i++)
{
for(j=0;j<n;j++)
 row_mat_copy[j]=row_mat[j];
        j=1;
        //this loop gives different class id to vertices with same class id but different state
        while(j<n)
        {
        if(map_g[ptr+j].classid==map_g[ptr+j-1].classid)
        {
         if(map_g[ptr+j].state!=map_g[ptr+j-1].state)
          {
            temp=map_g[ptr+j].classid;
          while(j<n && map_g[ptr+j].classid==temp){
            temps=map_g[ptr+j].state;
             while(j<n && map_g[ptr+j].state==temps && map_g[ptr+j].classid==temp){
              map_g[ptr+j].classid=classptr;
              j++;
              }
              classptr++;
              }  
             }  
          else j++;
             }
         else j++;
        }
               
        start=0;
        j=0;
        flag=false;
        while(j<n)
        {
        end=start+1;
        j++;
        while(j<n && map_g[ptr+end].classid==map_g[ptr+start].classid)
        {
          j++; end++;               
         }
        if(start<end-1){
           build_maxheap(row_mat,&map_g[ptr-1+start],end-start);
           heapsort(row_mat,&map_g[ptr-1+start],end-start); //subtracting 1 from array subscript for the padding for heap sort
 
        flag=true;
        }                        
        start=end;
        }
       
//writing state distribution vector to probability propogation matrix
for(j=0;j<n;j++)
 map_g[ptr+j].state=row_mat[map_g[ptr+j].map_ver];
 
//calculating the state distribution vector for string of next length
matrix_prod(row_mat,row_mat_copy,n,g,n,n);
}

start=0;
while(start<n-1){
 if(map_g[ptr+start].classid==map_g[ptr+start+1].classid)
  break;
 start++; }
end=start+1;
while(end<n-1){
 if(map_g[ptr+end].classid!=map_g[ptr+end+1].classid)
  break;
 end++; }
}
//returns the degree of a vertix
int degree(float *m,int row,int n)
{
int deg=0;
int base_ptr=row*n;
for(int i=0;i<n;i++){
deg+=(int)m[base_ptr+i];}
return deg;
}

//computing the probability distribution matrices
void prob_dibn(float *m,int n)
{
int deg;
for(int i=0;i<n;i++){
 deg = degree(m,i,n);
 for(int j=0;j<n;j++)
 {
 m[i*n+j]/=deg;
 }
}
}

void write(int graph_id,int initstate,mapping *map_g)
{
	char file_name[40];
	sprintf(file_name,"../graphiso/map_%d_%d",graph_id,initstate);
FILE *write = fopen(file_name,"w");
	for(int i=0;i<node;i++)
  fprintf(write,"%d ",map_g[i].map_ver);
 fprintf(write,"\n");   
fclose(write);
}

void get_graphs()
{
     int mode=0;
     char ch=' ';
     FILE *f = fopen("g1.txt","r");
      n1=0;n2=0;
         //checking the no: of nodes in the graph 1
		 while(ch!='\n')
         {
         ch = fgetc(f);
          if(ch>=48 && ch<=57 && mode==0)
           {
             mode=1;
               n1++;
            }
             else if(ch<48 || ch>57)
              mode=0;
          }
          node=n1;
          //initializing graph 1 and inputing values
         g1 = new float[n1*n1];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n1;i++)
         for(int j=0;j<n1;j++)
         fscanf(f,"%f ",&g1[i*n1+j]);
         fclose(f);
         ch=' ';
mode=0;
f = fopen("g2.txt","r");
//reading the adjacent matrix of Graph 2
//first checking the no: of elements in a row
while(ch!='\n')
{
ch = fgetc(f);
 if(ch>=48 && ch<=57 && mode==0)
 {
  mode=1;
  n2++;
 }
 else if(ch==' ')
  mode=0;
}
         g2 = new float[n2*n2];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n2;i++)
         for(int j=0;j<n2;j++)
         fscanf(f,"%f ",&g2[i*n2+j]);
         
         fclose(f); 
         
         //computing probability distribution matrices of both graphs
prob_dibn(g1,n1); //g1 is converted to the probability distribution matrix of graph 1
prob_dibn(g2,n2); //g2 is converted to the probability distribution matrix of graph 2          
     }

int main()
{
    #if defined(_WIN32)
    _mkdir("../graphiso");
    _mkdir("../results");
    #else
    mkdir("../graphiso", 0777);
    mkdir("../results",0777);
    #endif

FILE *result;

int pi,pj,iso=0;
char filename[40];

get_graphs();

FILE *read1,*read2;

if(n1==n2) //if number of vertices of both graphs are not equal then not isomorphic
{
	mapping *map = new mapping[n1*(n1+1)];
    mapping *m;
	float *graph1,*graph2,*rm,*rmc;
	hipMalloc((float**)&rm,sizeof(float)*n1);
	hipMalloc((float**)&rmc,sizeof(float)*n1);
	hipMalloc((float**)&graph1,sizeof(float)*n1*n1);
	hipMemcpy(graph1,g1,sizeof(float)*n1*n1,hipMemcpyHostToDevice);
	hipMalloc((float**)&graph2,sizeof(float)*n2*n2);
	hipMemcpy(graph2,g2,sizeof(float)*n2*n2,hipMemcpyHostToDevice);
 //dynamically allocating array for probability distribution matrix
	
   hipMalloc((mapping**)&m,sizeof(sizeof(mapping)*n1*(n1+1)));
 for(pi=0;(pi<n1)&&(iso!=2);pi++)
 {       
  sprintf(filename,"../graphiso/map_%d_%d",0,pi);  
  read1=fopen(filename,"r");
  if(!read1)
  {
   prob_prop_matrix<<<1,1>>>(0,g1,n1,m,rm,rmc);
   hipMemcpy(map,m,sizeof(mapping)*n1,hipMemcpyDeviceToHost);
   write(0,pi,map);
  }
   else
  fclose(read1);

   sprintf(filename,"../graphiso/map_%d_%d",1,0);
   read2=fopen(filename,"r");
   if(!read2)
   {
	   dim3 grids((n2+1)/2,1);
	   dim3 blocks(2,1);
    prob_prop_matrix<<<grids,blocks>>>(1,g2,n2,m,rm,rmc);
	 hipMemcpy(map,m,sizeof(mapping)*n1*(n1+1),hipMemcpyDeviceToHost);
	 for(int p=0;p<n2;p++)
	 write(1,p,&map[1+p]);
   }
    else
   fclose(read2);
 }

 hipFree(m);
  delete [] m;
  delete [] map;

  map_g = new mapping[2*n1];
   for(pi=0;(pi<n1)&&(iso!=2);pi++)
	for(pj=0;(pj<n2)&&(iso!=2);pj++)
		iso = isotest(pi,pj,g1,g2);
     
  if(iso==2)
{
sprintf(filename,"../results/res_%d_%d",pi,pj);
result=fopen(filename,"w");
fprintf(result,"ISOMORPHIC MAPPING\n");
for(int l=0;l<n1;l++)
fprintf(result,"%d -> %d\n",map_g[l].map_ver,map_g[n1+l].map_ver);
fprintf(result,"\n----------------\n");
fclose(result);
}
  }
else
cout<<"NOT ISOMORPHIC\n";

//deleting memory allocated for arrays
delete [] g1;
delete [] g2;
delete [] map_g;
getch();
return 0;
}

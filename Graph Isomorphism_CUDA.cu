#include "hip/hip_runtime.h"
// Copyright (c) 2014 Godly T.Alias
//
// This is a free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation,
// either version 3 of the License, or (at your option) any later version.

#include<iostream>
#include<conio.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
using namespace std;

int n1=0,n2=0;

struct mapping
{
int map_ver;
};

mapping *map_g;
float *gpu_g1,*gpu_g2,*gpu_p1;

//merge the partitions made by the mergesort
void merge(float *a,int s1,int e1,int s2,int e2,int mat)
{
int i,j,l,temp;
 if(s1<e2 && (e1+1)==s2)
 {
  i=s1;
  j=s2;
  while(i<=e1 && j<=e2)
  {
   l=0;
   while((a[l*n1+(map_g[mat*n1+i].map_ver)]==a[l*n1+(map_g[mat*n1+j].map_ver)]) && l<(n1-1))
   l++;
   if(a[l*n1+(map_g[mat*n1+i].map_ver)]<a[l*n1+(map_g[mat*n1+j].map_ver)])
   i++;
   else if(a[l*n1+(map_g[mat*n1+i].map_ver)]>=a[l*n1+(map_g[mat*n1+j].map_ver)])
    {
     temp = map_g[mat*n1+j].map_ver;
     for(int k=j;k>i;k--)
     map_g[mat*n1+k].map_ver=map_g[mat*n1+(k-1)].map_ver;
     map_g[mat*n1+i].map_ver=temp;
     j++;
     i++; e1++;
    }
  }
 }
}

//mergesort the matrix a and write the mapping to g_map[mat]
void mergesort(float *a,int start, int end,int mat)
{
if(start<(end-1))
 {
 mergesort(a,start,(start+end)/2,mat);
 mergesort(a,((start+end)/2)+1,end,mat);
 merge(a,start,(start+end)/2,((start+end)/2)+1,end,mat);
 }
else
 merge(a,start,start,end,end,mat);
}

//check whether the two matrices m1 & m2 are equal
__global__ void eq_matrix(float *m1,float *m2,mapping *gpu_map, bool *equal,int n)
{
//n1==n2
    int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;

if(i<(2*n-1) && j<n) {
   if(m1[i*n+gpu_map[j].map_ver]!=m2[i*n+gpu_map[n+j].map_ver])
	   *equal=false;}
}


__global__ void adj_mat_map(float *a1, float *a2,mapping *gpu_map,bool *equal,int n)
{
//n1==n2
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
if(i<n && j<n) {	
  if(a1[(gpu_map[i].map_ver*n)+gpu_map[j].map_ver]!=a2[(gpu_map[n+i].map_ver*n)+gpu_map[n+j].map_ver])
	  *equal=false;}
}

int isotest(float *p1,float *p2)
{
mergesort(p1,0,n1-1,0);
mergesort(p2,0,n2-1,1);
float *gpu_p2;
bool *gpu_equal,equal;
equal = true;
int sizeb = sizeof(bool);
int sizem = 2*n1*sizeof(mapping);
mapping *map;
int size = (2*n1-1)*n1*sizeof(float);

//allocating memory for gpu variables
hipMalloc((void**)&gpu_p2,size);
hipMalloc((void**)&map,sizem);
hipMalloc((void**)&gpu_equal,sizeb);
//copying to gpu variables
hipMemcpy(gpu_p2,p2,size,hipMemcpyHostToDevice);
hipMemcpy(gpu_equal,&equal,sizeb,hipMemcpyHostToDevice);
hipMemcpy(map,map_g,sizem,hipMemcpyHostToDevice);

eq_matrix<<<dim3(n1/2 , (2*n1-1)/2), dim3(2,2)>>>(gpu_p1,gpu_p2,map,gpu_equal,n1);

hipMemcpy(&equal,gpu_equal,sizeb,hipMemcpyDeviceToHost);
hipFree(gpu_p2);
if(equal)
 {
	 equal=true;
	 //copying to gpu variables
	 hipMemcpy(gpu_equal,&equal,sizeb,hipMemcpyHostToDevice);

	 adj_mat_map<<<dim3(n1/2,n1/2),dim3(2,2)>>>(gpu_g1,gpu_g2,map,gpu_equal,n1);

	 hipMemcpy(&equal,gpu_equal,sizeb,hipMemcpyDeviceToHost);
	 hipFree(gpu_equal); hipFree(map);
  if(equal)
   return 2;
  else
   return 1;
 }
else
 return 0;
}

//returns the initial state distribution vector
void istate_dibn_vec(float* init_state, int i,int n)
{
for(int j=0;j<n;j++)
 if(j==i)
  init_state[j]=1.0;
 else
  init_state[j]=0.0;
}

//computes the product of matrices m1 & m2 and write the result in res matrix
__global__ void matrix_prod(float *res,float *m1,int c1,float *m2,int r2,int c2)
{
float y,t,c;
int j=blockIdx.x*blockDim.x+threadIdx.x;
if(c1==r2 && j<c2){
 res[j]=0;
 c=0.0;
  for(int k=0;k<c1;k++){
//kahan summation to avoid precision lose
  y=(m1[k]*m2[k*c2+j])-c;
  t=res[j]+y;
  c = (t-res[j])-y;
  res[j]=t;}
}}

//calculates the probability propogation matrix for the initial state initstate
float* prob_prop_matrix(float *p, float *gpu, int n, int initstate)
{
float *row_mat,*gpu_rowmat,*row_mat_copy;
//dynamically allocating array for probability distribution matrix
p = new float[n*((2*n)-1)];
//row_mat holds the value of each state distribution vector
row_mat = new float[n];
//writes the initial state vector to the row_mat
istate_dibn_vec(row_mat,initstate,n);
int size = n*sizeof(float);
dim3 blocksize(2,1);
dim3 gridsize((n/blocksize.x)+((n/blocksize.x)%n),1);
hipMalloc((void**)&gpu_rowmat,size);
hipMalloc((void**)&row_mat_copy,size);

for(int i=0;i<((2*n)-1);i++)
{
//copying state distribution vector to probability propogation matrix
 for(int j=0;j<n;j++)
  p[i*n +j]=row_mat[j];

 hipMemcpy(row_mat_copy,row_mat,size,hipMemcpyHostToDevice);
//calculating the state distribution vector for string of next length
 matrix_prod<<<gridsize,blocksize>>>(gpu_rowmat,row_mat_copy,n1,gpu,n1,n1); //n1==n2
 hipMemcpy(row_mat,gpu_rowmat,size,hipMemcpyDeviceToHost);
}

//deleting the allocated memory
hipFree(gpu_rowmat);
hipFree(row_mat_copy);
delete [] row_mat;
return p;
}

//returns the degree of a vertix
__device__ int degree(float *m,int row,int n)
{
int deg=0;
for(int i=0;i<n;i++)
deg+=m[row*n+i];
return deg;
}

//computing the probability distribution matrices
__global__ void prob_dibn(float *m,int n)
{
int deg;
int y=blockIdx.y*blockDim.y+threadIdx.y;
if(y<n){
 deg = degree(m,y,n);
for(int i=0;i<n;i++)
	m[y*n+i]/=deg;}
}

int main()
{
int i,j,mode,pi,pj,iso=0,psize,gsize;
float *p1=NULL,*p2=NULL,*g1,*g2;
char ch;
ch=' ';
mode=0;

FILE *read1 = fopen("g1.txt","r");
//reading the adjacent matrix of Graph 1
//first checking the no: of elements in a row
while(ch!='\n')
{
ch = fgetc(read1);
 if(ch>=48 && ch<=57 && mode==0)
 {
  mode=1;
  n1++;
 }
 else if(ch<48 || ch>57)
  mode=0;
}

//dynamically allocating array
map_g = new mapping[2*n1];

g1 = new float[n1*n1];
for(i=0;i<n1;i++){
map_g[i].map_ver=i;
map_g[n1+i].map_ver=i;
}

fseek(read1,0,SEEK_SET);
for(i=0;i<n1;i++)
for(j=0;j<n1;j++){
fscanf(read1,"%f",&g1[i*n1+j]);
}

fclose(read1);
ch=' ';
mode=0;
FILE *read2 = fopen("g2.txt","r");
//reading the adjacent matrix of Graph 2
//first checking the no: of elements in a row
while(ch!='\n')
{
ch = fgetc(read2);
 if(ch>=48 && ch<=57 && mode==0)
 {
  mode=1;
  n2++;
 }
 else if(ch==' ')
  mode=0;
}

//dynamically allocating array
g2 = new float[n2*n2];

fseek(read2,0,SEEK_SET);
for(i=0;i<n2;i++)
for(j=0;j<n2;j++){
fscanf(read2,"%f",&g2[i*n2 +j]);
}

fclose(read2);

if(n1==n2) //if number of vertices of both graphs are not equal then not isomorphic
{
 dim3 dimBlock(1,1);
 dim3 dimGrid(1,n1);
 gsize = n1*n1*sizeof(float);
//computing probability distribution matrices of both graphs
 hipMalloc((void**)&gpu_g1,gsize);
 hipMemcpy(gpu_g1,g1,gsize,hipMemcpyHostToDevice);
 prob_dibn<<<dimGrid,dimBlock>>>(gpu_g1,n1); //g1 is converted to the probability distribution matrix of graph 1
 
 
 hipMalloc((void**)&gpu_g2,gsize);
 hipMemcpy(gpu_g2,g2,gsize,hipMemcpyHostToDevice);
 prob_dibn<<<dimGrid,dimBlock>>>(gpu_g2,n2); //g2 is converted to the probability distribution matrix of graph 2
 
    delete [] g1;
	delete [] g2;

 iso=0;
  psize = (2*n1-1)*n1*sizeof(float);
 for(pi=0;(pi<n1)&&(iso!=2);pi++)
 {
  p1=prob_prop_matrix(p1,gpu_g1,n1,pi);  
  hipMalloc((void**)&gpu_p1,psize);
  hipMemcpy(gpu_p1,p1,psize,hipMemcpyHostToDevice);
  
  for(pj=0;(pj<n2)&&(iso!=2);pj++)
  {
   p2=prob_prop_matrix(p2,gpu_g2,n2,pj);
   iso = isotest(p1,p2);
   //deleting the memory for the probability propogation matrix
       delete [] p2;
  }
  //deleting the memory for probability propogation matrix
  delete [] p1;
  hipFree(gpu_p1);
 }
}
else
 iso=0;

if(iso==2)
{
cout<<"ISOMORPHIC MAPPING :\n";
for(i=0;i<n1;i++)
cout<<map_g[i].map_ver<<"->"<<map_g[n1+i].map_ver<<"\n";
}
else
cout<<"NOT ISOMORPHIC\n";
//deleting memory allocated for arrays
	delete [] map_g;
	hipFree(gpu_g1);
    hipFree(gpu_g2);
getch();
return 0;
}

// Copyright (c) 2014 Godly T.Alias
//
// This is a free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation,
// either version 3 of the License, or (at your option) any later version.

#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include<time.h>

#include<sys/stat.h>
using namespace std;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        FILE *error = fopen("GPU_error.txt","w");
        fprintf(error, "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
	fclose(error);
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int n1,n2,perm;

struct mapping
{
int map_ver;
float state;
int classid; };


float *g1,*g2;
mapping *map_graph;
int node,w_node;
int tmp_count;

//max heapify part of heap sort
__device__ void max_heapify(float *a,mapping *pos, int i, int n)
{
    int j, temps;
    float temp;
    temps = pos[i].map_ver;
    temp = a[temps];
    j = 2*i;
    while (j <= n)
    {
        if (j < n && a[pos[j+1].map_ver] > a[pos[j].map_ver])
            j = j+1;
        if (temp > a[pos[j].map_ver])
            break;
        else if (temp <= a[pos[j].map_ver])
        {
            pos[j/2].map_ver = pos[j].map_ver;
            j = 2*j;
        }
    }
    pos[j/2].map_ver = temps;
    return;
}

//sorts the given array a and stores the mappings in pos
__device__ void heapsort(float *a,mapping *pos, int end)
{
    int i, temps;
    for (i = end; i >= 2; i--)
    {
        temps = pos[i].map_ver;
        pos[i].map_ver = pos[1].map_ver;
        pos[1].map_ver = temps;
        max_heapify(a,pos, 1, i - 1);
    }
}

__device__ void build_maxheap(float *a,mapping *pos, int end)
{
    int i;
    for(i = end/2; i >= 1; i--)
    {
        max_heapify(a,pos, i, end);
    }
}

//check whether map1(a1) == map2(a2) for all elements
__device__ bool adj_mat_map(float *a1, float *a2,mapping *map1,mapping *map2,int n)
{
int i,j;
for(i=0;i<n;i++)
    for(j=0;j<n;j++)
        if(a1[map1[i].map_ver*n+map1[j].map_ver]!=a2[map2[i].map_ver*n+map2[j].map_ver])
            return false;
return true;
}

//check whether both the probability propogation matrices are isomorphic and
//if isomorphic put the number of the initial node to the variable isonode
__global__ void isotest(float *a1,float *a2,mapping *map1,mapping *map2,int *isonode,int n)
{
	int id = threadIdx.x+blockIdx.x*blockDim.x;
 if(id<n)
     if(adj_mat_map(a1,a2,map1,&map2[id*n],n))
         *isonode=id;
}


//returns the initial state distribution vector
__device__ void istate_dibn_vec(float* init_state, int i,int n)
{
for(int j=0;j<n;j++)
 if(j==i)
  init_state[j]=1.0;
 else
  init_state[j]=0.0;
}

//computes the product of matrices m1 & m2 and write the result in res matrix
__device__ void matrix_prod(float *res,float *m1,int c1,float *m2,int r2,int c2)
{
float y,t,c;
if(c1==r2){
    for(int j=0;j<c2;j++){
        res[j]=0;
        c=0.0;
        for(int k=0;k<c1;k++){
//kahan summation to avoid precision lose
            y=(m1[k]*m2[k*c2+j])-c;
            t=res[j]+y;
            c = (t-res[j])-y;
            res[j]=t;}
        }
    }
}

//calculates the probability propogation matrix for the initial state initstate
__global__ void prob_prop_matrix(int graph_id, float *g, int n, mapping *map_g,float *rm,float *rmc)
{
	float *row_mat,*row_mat_copy;
	int initstate = blockIdx.x*blockDim.x+threadIdx.x;
if(initstate<n){
    int ptr = initstate*n;
    for(int i=0;i<n;i++)
        {
        map_g[ptr+i].map_ver=i;
        map_g[ptr+i].state=-1.0;
        map_g[ptr+i].classid=0;
        }
    row_mat = &rm[ptr];
    row_mat_copy = &rmc[ptr];
	 
bool flag=true;
int start,end,j,temp,classptr;
float temps;

//writes the initial state vector to the row_mat
istate_dibn_vec(row_mat,initstate,n);
classptr=1;
for(int i=0;flag && i<((2*n)-1);i++)
{
    for(j=0;j<n;j++)
        row_mat_copy[j]=row_mat[j];
    j=1;
        //this loop gives different class id to vertices with same class id but different state
    while(j<n)
        {
        if(map_g[ptr+j].classid==map_g[ptr+j-1].classid)
        {
         if(map_g[ptr+j].state!=map_g[ptr+j-1].state)
          {
            temp=map_g[ptr+j].classid;
          while(j<n && map_g[ptr+j].classid==temp){
            temps=map_g[ptr+j].state;
             while(j<n && map_g[ptr+j].state==temps && map_g[ptr+j].classid==temp){
              map_g[ptr+j].classid=classptr;
              j++;
              }
              classptr++;
              }  
             }  
          else j++;
             }
         else j++;
        }
               
        start=0;
        j=0;
        flag=false;
        while(j<n)
        {
            end=start+1;
            j++;
            while(j<n && map_g[ptr+end].classid==map_g[ptr+start].classid)
                {
                j++; end++;               
                }
            if(start<end-1){
               build_maxheap(row_mat,&map_g[ptr-1+start],end-start);
               heapsort(row_mat,&map_g[ptr-1+start],end-start); //subtracting 1 from array subscript for the padding for heap sort
               flag=true;
            }                        
            start=end;
        }
       
//writing state distribution vector to probability propogation matrix
for(j=0;j<n;j++)
 map_g[ptr+j].state=row_mat[map_g[ptr+j].map_ver];
 
//calculating the state distribution vector for string of next length
matrix_prod(row_mat,row_mat_copy,n,g,n,n);
}
}
}


//returns the degree of a vertix
int degree(float *m,int row,int n)
{
int deg=0;
int base_ptr=row*n;
for(int i=0;i<n;i++){
    deg+=(int)m[base_ptr+i];}
return deg;
}

//computing the probability distribution matrices
void prob_dibn(float *m,int n)
{
int deg;
for(int i=0;i<n;i++){
 deg = degree(m,i,n);
  for(int j=0;j<n;j++)
  {
   m[i*n+j]/=deg;
  }
 }
}

//writes the mappings to file
void write(int graph_id,int initstate,mapping *map_g)
{
	char file_name[40];
	sprintf(file_name,"../graphiso/map_%d_%d",graph_id,initstate);
FILE *write = fopen(file_name,"w");
 for(int i=0;i<node;i++)
     fprintf(write,"%d ",map_g[i].map_ver);
 fprintf(write,"\n");   
 fclose(write);
}

void get_graphs()
{
     int mode=0;
     char ch=' ';
     FILE *f = fopen("g1.txt","r");
      n1=0;n2=0;
         //checking the no: of nodes in the graph 1
    while(ch!='\n')
    {
        ch = fgetc(f);
        if(ch>=48 && ch<=57 && mode==0)
        {
            mode=1;
            n1++;
        }
        else if(ch<48 || ch>57)
            mode=0;
    }
          node=n1;
          //initializing graph 1 and inputing values
         g1 = new float[n1*n1];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n1;i++)
             for(int j=0;j<n1;j++)
                 fscanf(f,"%f ",&g1[i*n1+j]);
         fclose(f);
         ch=' ';
mode=0;
f = fopen("g2.txt","r");
//reading the adjacent matrix of Graph 2
//first checking the no: of elements in a row
while(ch!='\n')
{
    ch = fgetc(f);
    if(ch>=48 && ch<=57 && mode==0)
    {
        mode=1;
        n2++;
    }
    else if(ch==' ')
        mode=0;
}
         g2 = new float[n2*n2];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n2;i++)
             for(int j=0;j<n2;j++)
                 fscanf(f,"%f ",&g2[i*n2+j]);
         
         fclose(f); 
         
         //computing probability distribution matrices of both graphs
prob_dibn(g1,n1); //g1 is converted to the probability distribution matrix of graph 1
prob_dibn(g2,n2); //g2 is converted to the probability distribution matrix of graph 2          
}

int main()
{
	time_t start = time(0);
    #if defined(_WIN32)
    _mkdir("../graphiso");
    _mkdir("../results");
    #else
    mkdir("../graphiso", 0777);
    mkdir("../results",0777);
    #endif

FILE *result;
float *graph1,*graph2,*rm,*rmc;

int *iso,ison;
char filename[40];
int *isonode;
mapping *m;

//get the graphs from the respective files and 
//calculate its probability distribution matrix
get_graphs();

mapping *map;
FILE *read1,*read2;

if(n1==n2) //if number of vertices of both graphs are not equal then not isomorphic
{
	map = new mapping[n1*n1];
	HANDLE_ERROR(hipMalloc((float**)&rm,sizeof(float)*n1*n1));  //for copying each row of probability propogation matrix
	HANDLE_ERROR(hipMalloc((float**)&rmc,sizeof(float)*n1*n1)); //copy of rm for getting matrix product with each row
	HANDLE_ERROR(hipMalloc((float**)&graph1,sizeof(float)*n1*n1)); //adjacency matrix of 1st graph
	HANDLE_ERROR(hipMemcpy(graph1,g1,sizeof(float)*n1*n1,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((float**)&graph2,sizeof(float)*n2*n2)); //adjacency matrix of 2nd graph
	HANDLE_ERROR(hipMemcpy(graph2,g2,sizeof(float)*n2*n2,hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMalloc((mapping**)&m,sizeof(mapping)*n1*n1));
       
//after allocating memory in GPU graphs are no
//more to be stored in the RAM
delete [] g1;
delete [] g2;

  sprintf(filename,"../graphiso/map_%d_%d",0,0);  
  read1=fopen(filename,"r");
  if(!read1) //checks whether the mappings had already been created
  {
	   dim3 grids((n1+1)/2,1);
	   dim3 blocks(2,1);
  
   prob_prop_matrix<<<grids,blocks>>>(0,graph1,n1,m,rm,rmc); //invokes the kernel for finding probability
   							     //propogation matrices for 1st graph
   
   HANDLE_ERROR( hipPeekAtLastError() );
   HANDLE_ERROR(hipMemcpy(map,m,sizeof(mapping)*n1*n1,hipMemcpyDeviceToHost));
   
	 for(int p=0;p<n1;p++) //writes the mappings to file
	     write(0,p,&map[p*n1]);
  }
   else
    fclose(read1);

   sprintf(filename,"../graphiso/map_%d_%d",1,0);
   read2=fopen(filename,"r");
   if(!read2) //checks whether the mappings had already been created
   {
	   dim3 grids((n2+1)/2,1);
	   dim3 blocks(2,1);
   
    prob_prop_matrix<<<grids,blocks>>>(1,graph2,n2,m,rm,rmc);
   
         HANDLE_ERROR( hipPeekAtLastError() );
	 HANDLE_ERROR(hipMemcpy(map,m,sizeof(mapping)*n2*n2,hipMemcpyDeviceToHost));
     for(int p=0;p<n2;p++)
	 write(1,p,&map[p*n2]);
   }
   else
       fclose(read2);

 hipFree(rm);
 hipFree(rmc);

 mapping *m_g1;
 map_graph = new mapping[node];
	   HANDLE_ERROR(hipMalloc((mapping**)&m_g1,sizeof(mapping)*n1));

 ison=-1;
 iso=&ison;

 for(int pi=0;(pi<n1)&&(*iso<0);pi++){
     sprintf(filename,"../graphiso/map_%d_%d",0,pi);
     read1=fopen(filename,"r");
     for(int i=0;i<n1;i++)
         fscanf(read1,"%d ",&map_graph[i].map_ver);
     fclose(read1);
     //copies the mapping of pi-th node of graph 1 to GPU
	   HANDLE_ERROR(hipMemcpy(m_g1,map_graph,sizeof(mapping)*n1,hipMemcpyHostToDevice));
	   HANDLE_ERROR(hipMalloc((int**)&isonode,sizeof(int)));
	   HANDLE_ERROR(hipMemcpy(isonode,iso,sizeof(int),hipMemcpyHostToDevice));
     dim3 grids((n1+1)/2,1);
     dim3 threads(2,1);
	//checks whether there is an isomorphic mapping
		isotest<<<grids,threads>>>(graph1,graph2,m_g1,m,isonode,node);     
          
          HANDLE_ERROR( hipPeekAtLastError() );
	  HANDLE_ERROR(hipMemcpy(iso,isonode,sizeof(int),hipMemcpyDeviceToHost));
	  HANDLE_ERROR(hipFree(isonode));
	  
     if(ison>=0)
     {
         sprintf(filename,"../results/res_%d_%d",pi,ison);
         result=fopen(filename,"w");
         fprintf(result,"ISOMORPHIC MAPPING\n");
         for(int l=0;l<n1;l++)
             fprintf(result,"%d -> %d\n",map_graph[l].map_ver,map[(ison*node)+l].map_ver);
         fprintf(result,"\n----------------\n");
         fclose(result);
     }
   }

   	hipFree(m_g1);
}
if(*iso<0)
    cout<<"NOT ISOMORPHIC\n";

//deleting memory allocated for arrays
hipFree(graph1);
hipFree(graph2);
hipFree(m);
delete [] map;
delete [] map_graph;
time_t end = time(0);
cout<<"Time taken - "<<end-start;
return 0;
}

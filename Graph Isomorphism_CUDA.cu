#include "hip/hip_runtime.h"
// Copyright (c) 2014 Godly T.Alias
//
// This is a free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation,
// either version 3 of the License, or (at your option) any later version.

#include<iostream>
#include<conio.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<direct.h>
using namespace std;
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
		FILE *error = fopen("GPU_error.txt","w");
        fprintf(error, "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		fclose(error);
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int n1,n2,perm;
struct mapping
{
int map_ver;
float state;
int classid; };


float *g1,*g2;
mapping *map_graph;
int node,w_node;
int tmp_count;

__device__ void max_heapify(float *a,mapping *pos, int i, int n)
{
    int j, temps;
    float temp;
    temps = pos[i].map_ver;
    temp = a[temps];
    j = 2*i;
    while (j <= n)
    {
        if (j < n && a[pos[j+1].map_ver] > a[pos[j].map_ver])
            j = j+1;
        if (temp > a[pos[j].map_ver])
            break;
        else if (temp <= a[pos[j].map_ver])
        {
            pos[j/2].map_ver = pos[j].map_ver;
            j = 2*j;
        }
    }
    pos[j/2].map_ver = temps;
    return;
}

__device__ void heapsort(float *a,mapping *pos, int end)
{
    int i, temps;
    for (i = end; i >= 2; i--)
    {
        temps = pos[i].map_ver;
        pos[i].map_ver = pos[1].map_ver;
        pos[1].map_ver = temps;
        max_heapify(a,pos, 1, i - 1);
    }
}

__device__ void build_maxheap(float *a,mapping *pos, int end)
{
    int i;
    for(i = end/2; i >= 1; i--)
    {
        max_heapify(a,pos, i, end);
    }
}

__global__ void adj_mat_map(float *a1, float *a2,mapping *map_g1,mapping *map_g2,bool *iso,int init, int node)
{
 int i=blockIdx.x*blockDim.x+threadIdx.x;
 int j=blockIdx.y*blockDim.y+threadIdx.y;
 if(i<node && j<node){
  if(a1[map_g1[i].map_ver*node+map_g1[j].map_ver]!=a2[map_g2[i].map_ver*node+map_g2[j].map_ver])
  *iso=false;
 }}


int isotest(int p2_init_node,float *a1,float *a2,mapping *map_g1,mapping *map)
{
	bool *is_iso,*iso;
	bool val = true;
	is_iso = &val;
	mapping *map_g2;

HANDLE_ERROR(hipMalloc((bool**)&iso,sizeof(bool)));
HANDLE_ERROR(hipMalloc((mapping**)&map_g2,sizeof(mapping)*n1));
HANDLE_ERROR(hipMemcpy(map_g2,map,sizeof(mapping)*n1,hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(iso,is_iso,sizeof(bool),hipMemcpyHostToDevice));

dim3 threadsPerblock(2,2);
dim3 blocks((node+1)/2,(node+1)/2);

adj_mat_map<<<blocks,threadsPerblock>>>(a1,a2,map_g1,map_g2,iso,p2_init_node,node);

HANDLE_ERROR(hipMemcpy(is_iso,iso,sizeof(bool),hipMemcpyDeviceToHost));
hipFree(iso);
hipFree(map_g2);
  if(*is_iso)
	return 2;
  else
   return 0;
}


//returns the initial state distribution vector
__device__ void istate_dibn_vec(float* init_state, int i,int n)
{
for(int j=0;j<n;j++)
 if(j==i)
  init_state[j]=1.0;
 else
  init_state[j]=0.0;
}

//computes the product of matrices m1 & m2 and write the result in res matrix
__device__ void matrix_prod(float *res,float *m1,int c1,float *m2,int r2,int c2)
{
float y,t,c;
if(c1==r2){
 for(int j=0;j<c2;j++){
 res[j]=0;
 c=0.0;
  for(int k=0;k<c1;k++){
//kahan summation to avoid precision lose
  y=(m1[k]*m2[k*c2+j])-c;
  t=res[j]+y;
  c = (t-res[j])-y;
  res[j]=t;}
 }
}
}

//calculates the probability propogation matrix for the initial state initstate
__global__ void prob_prop_matrix(int graph_id, float *g, int n, mapping *map_g,float *rm,float *rmc)
{
	float *row_mat,*row_mat_copy;
	int initstate = blockIdx.x*blockDim.x+threadIdx.x;
if(initstate<n){
	int ptr = initstate*n;
	 for(int i=0;i<n;i++)
        {
        map_g[ptr+i].map_ver=i;
        map_g[ptr+i].state=-1.0;
        map_g[ptr+i].classid=0;
        }
	 row_mat = &rm[ptr];
	 row_mat_copy = &rmc[ptr];
	 
bool flag=true;
int start,end,j,temp,classptr;
float temps;

//writes the initial state vector to the row_mat
istate_dibn_vec(row_mat,initstate,n);
classptr=1;
for(int i=0;flag && i<((2*n)-1);i++)
{
for(j=0;j<n;j++)
 row_mat_copy[j]=row_mat[j];
        j=1;
        //this loop gives different class id to vertices with same class id but different state
        while(j<n)
        {
        if(map_g[ptr+j].classid==map_g[ptr+j-1].classid)
        {
         if(map_g[ptr+j].state!=map_g[ptr+j-1].state)
          {
            temp=map_g[ptr+j].classid;
          while(j<n && map_g[ptr+j].classid==temp){
            temps=map_g[ptr+j].state;
             while(j<n && map_g[ptr+j].state==temps && map_g[ptr+j].classid==temp){
              map_g[ptr+j].classid=classptr;
              j++;
              }
              classptr++;
              }  
             }  
          else j++;
             }
         else j++;
        }
               
        start=0;
        j=0;
        flag=false;
        while(j<n)
        {
        end=start+1;
        j++;
        while(j<n && map_g[ptr+end].classid==map_g[ptr+start].classid)
        {
          j++; end++;               
         }
        if(start<end-1){
           build_maxheap(row_mat,&map_g[ptr-1+start],end-start);
           heapsort(row_mat,&map_g[ptr-1+start],end-start); //subtracting 1 from array subscript for the padding for heap sort
 
        flag=true;
        }                        
        start=end;
        }
       
//writing state distribution vector to probability propogation matrix
for(j=0;j<n;j++)
 map_g[ptr+j].state=row_mat[map_g[ptr+j].map_ver];
 
//calculating the state distribution vector for string of next length
matrix_prod(row_mat,row_mat_copy,n,g,n,n);
}
}
}
//returns the degree of a vertix
int degree(float *m,int row,int n)
{
int deg=0;
int base_ptr=row*n;
for(int i=0;i<n;i++){
deg+=(int)m[base_ptr+i];}
return deg;
}

//computing the probability distribution matrices
void prob_dibn(float *m,int n)
{
int deg;
for(int i=0;i<n;i++){
 deg = degree(m,i,n);
 for(int j=0;j<n;j++)
 {
 m[i*n+j]/=deg;
 }
}
}

void write(int graph_id,int initstate,mapping *map_g)
{
	char file_name[40];
	sprintf(file_name,"../graphiso/map_%d_%d",graph_id,initstate);
FILE *write = fopen(file_name,"w");
	for(int i=0;i<node;i++)
  fprintf(write,"%d ",map_g[i].map_ver);
 fprintf(write,"\n");   
fclose(write);
}

void get_graphs()
{
     int mode=0;
     char ch=' ';
     FILE *f = fopen("g1.txt","r");
      n1=0;n2=0;
         //checking the no: of nodes in the graph 1
		 while(ch!='\n')
         {
         ch = fgetc(f);
          if(ch>=48 && ch<=57 && mode==0)
           {
             mode=1;
               n1++;
            }
             else if(ch<48 || ch>57)
              mode=0;
          }
          node=n1;
          //initializing graph 1 and inputing values
         g1 = new float[n1*n1];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n1;i++)
         for(int j=0;j<n1;j++)
         fscanf(f,"%f ",&g1[i*n1+j]);
         fclose(f);
         ch=' ';
mode=0;
f = fopen("g2.txt","r");
//reading the adjacent matrix of Graph 2
//first checking the no: of elements in a row
while(ch!='\n')
{
ch = fgetc(f);
 if(ch>=48 && ch<=57 && mode==0)
 {
  mode=1;
  n2++;
 }
 else if(ch==' ')
  mode=0;
}
         g2 = new float[n2*n2];
         fseek(f,0,SEEK_SET);
         for(int i=0;i<n2;i++)
         for(int j=0;j<n2;j++)
         fscanf(f,"%f ",&g2[i*n2+j]);
         
         fclose(f); 
         
         //computing probability distribution matrices of both graphs
prob_dibn(g1,n1); //g1 is converted to the probability distribution matrix of graph 1
prob_dibn(g2,n2); //g2 is converted to the probability distribution matrix of graph 2          
     }

int main()
{
    #if defined(_WIN32)
    _mkdir("../graphiso");
    _mkdir("../results");
    #else
    mkdir("../graphiso", 0777);
    mkdir("../results",0777);
    #endif

FILE *result;
	float *graph1,*graph2,*rm,*rmc;

int iso=0;
char filename[40];

get_graphs();

mapping *map;
FILE *read1,*read2;

if(n1==n2) //if number of vertices of both graphs are not equal then not isomorphic
{
	map = new mapping[n1*n1];
    mapping *m;
	HANDLE_ERROR(hipMalloc((float**)&rm,sizeof(float)*n1*n1));
	HANDLE_ERROR(hipMalloc((float**)&rmc,sizeof(float)*n1*n1));
	HANDLE_ERROR(hipMalloc((float**)&graph1,sizeof(float)*n1*n1));
	HANDLE_ERROR(hipMemcpy(graph1,g1,sizeof(float)*n1*n1,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((float**)&graph2,sizeof(float)*n2*n2));
	HANDLE_ERROR(hipMemcpy(graph2,g2,sizeof(float)*n2*n2,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((mapping**)&m,sizeof(mapping)*n1*n1));
       
  sprintf(filename,"../graphiso/map_%d_%d",0,0);  
  read1=fopen(filename,"r");
  if(!read1)
  {
	   dim3 grids((n1+1)/2,1);
	   dim3 blocks(2,1);
   prob_prop_matrix<<<grids,blocks>>>(0,graph1,n1,m,rm,rmc);
   HANDLE_ERROR( hipPeekAtLastError() );
   HANDLE_ERROR(hipMemcpy(map,m,sizeof(mapping)*n1*n1,hipMemcpyDeviceToHost));
	 for(int p=0;p<n1;p++)
	 write(0,p,&map[p*n1]);
  }
   else
  fclose(read1);

   sprintf(filename,"../graphiso/map_%d_%d",1,0);
   read2=fopen(filename,"r");
   if(!read2)
   {
	   dim3 grids((n2+1)/2,1);
	   dim3 blocks(2,1);
    prob_prop_matrix<<<grids,blocks>>>(1,graph2,n2,m,rm,rmc);
	 HANDLE_ERROR(hipMemcpy(map,m,sizeof(mapping)*n2*n2,hipMemcpyDeviceToHost));
	 for(int p=0;p<n2;p++)
	 write(1,p,&map[p*n2]);
   }
    else
   fclose(read2);

 hipFree(m);
 hipFree(rm);
 hipFree(rmc);

 mapping *m_g1;
  map_graph = new mapping[node];
	   HANDLE_ERROR(hipMalloc((mapping**)&m_g1,sizeof(mapping)*n1));

   for(int pi=0;(pi<n1)&&(iso!=2);pi++){
	   sprintf(filename,"../graphiso/map_%d_%d",0,pi);
	   read1=fopen(filename,"r");
	   for(int i=0;i<n1;i++)
		   fscanf(read1,"%d ",&map_graph[i].map_ver);
           fclose(read1);
	   hipMemcpy(m_g1,map_graph,sizeof(mapping)*n1,hipMemcpyHostToDevice);
	for(int pj=0;(pj<n2)&&(iso!=2);pj++){
		iso = isotest(pj,graph1,graph2,m_g1,&map[pj*node]);     
	 if(iso==2)
	{
	sprintf(filename,"../results/res_%d_%d",pi,pj);
	result=fopen(filename,"w");
	fprintf(result,"ISOMORPHIC MAPPING\n");
	for(int l=0;l<n1;l++)
		fprintf(result,"%d -> %d\n",map_graph[l].map_ver,map[(pj*node)+l].map_ver);
	fprintf(result,"\n----------------\n");
	fclose(result);
	}
	}
   }
   	hipFree(m_g1);
}

if(iso!=2)
cout<<"NOT ISOMORPHIC\n";

//deleting memory allocated for arrays
delete [] g1;
delete [] g2;
hipFree(graph1);
hipFree(graph2);
delete [] map;
delete [] map_graph;
getch();
return 0;
}
